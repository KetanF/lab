
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Sequential vector addition function
void vectorAddSequential(int* a, int* b, int* c, int size) {
    for (int i = 0; i < size; i++) {
        c[i] = a[i] + b[i];
    }
}

// CUDA kernel for vector addition
__global__ void vectorAddParallel(int* a, int* b, int* c, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int main() {
    int size;
    printf("Enter the size of the vectors: ");
    scanf("%d", &size);

    // Allocate memory for host vectors
    int* a = (int*)malloc(size * sizeof(int));
    int* b = (int*)malloc(size * sizeof(int));
    int* c = (int*)malloc(size * sizeof(int));

    // Initialize host vectors
    printf("Enter elements for vector a:\n");
    for (int i = 0; i < size; i++) {
        scanf("%d", &a[i]);
    }
    printf("Enter elements for vector b:\n");
    for (int i = 0; i < size; i++) {
        scanf("%d", &b[i]);
    }

    // Allocate memory on the device for device vectors
    int* dev_a, * dev_b, * dev_c;
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));
    hipMalloc((void**)&dev_c, size * sizeof(int));

    // Copy host vectors to device
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Parallel vector addition
    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    clock_t start_parallel = clock();
    vectorAddParallel<<<gridSize, blockSize>>>(dev_a, dev_b, dev_c, size);
    hipDeviceSynchronize(); // Wait for the GPU to finish
    clock_t end_parallel = clock();
    double time_parallel = ((double)(end_parallel - start_parallel)) / CLOCKS_PER_SEC;

    // Copy result from device to host
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    // Sequential vector addition
    clock_t start_sequential = clock();
    vectorAddSequential(a, b, c, size);
    clock_t end_sequential = clock();
    double time_sequential = ((double)(end_sequential - start_sequential)) / CLOCKS_PER_SEC;

    // Print result
    printf("\nResult of Parallel Vector Addition:\n");
    for (int i = 0; i < size; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    printf("\nTime taken for Parallel Vector Addition: %lf seconds\n", time_parallel);
    printf("Time taken for Sequential Vector Addition: %lf seconds\n", time_sequential);

    // Free device memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    // Free host memory
    free(a);
    free(b);
    free(c);

    return 0;
}
